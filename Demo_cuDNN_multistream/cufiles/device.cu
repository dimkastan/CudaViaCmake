#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>


#include <chrono>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

/*
Description:
 
 This is a self contained demo, showing the usage of cuDNN Application Programming Interface (API)
 For simplicity in [1], we define the size of a convolution filter (width, height and number of filters) as well as the size of the image (batch size, input channels, width, height)
 and next we allocate all cudnn-specific tensors and allocate cuda memory in order to perform forward passes.

Currently it is not tested, but is should be ok once you add artificial data inside weights and image.

Author: Dimitris Kastaniotis, November 2017
 
*/

 
using namespace std;
 
 
int ITERS = 1000;  // Measure time as an average of 1000 Iterations.

	
hipdnnStatus_t err;
#define CHECK_CUDNN(err) if ((err) != HIPDNN_STATUS_SUCCESS) {  printf("[%s: %i]: CUDNN Error: %d %s\n",__FILE__,__LINE__, err, printf("Error code: %d\n",err)); return -1; }

int cudnn_function(hipdnnHandle_t &cudnnHandle, hipStream_t &cudaStream ){
	
	int kernel_size;                         //holds the  size of the convolution kernel (only symmetric kernels are supported here)
	int NumberOfOutputFilters;               // holds the number of output channels
	int NumberOfInputFilters;                // holds the number of input channels
	int stepsize;                            // holds the step size
	int padd;                                // holds the padding size
	int dilation_h, dilation_w;              // holds the dlation size
	
	int    in_width=0;              // input image/feature map width
	int    in_height=0;             // input image/feature map height
	int    out_height=0;            // output image/feature map width
	int    out_width=0;             // output image/feature map height
	int    in_channels=0;           // input image/feature map channels
	int    out_channels=0;          // output image/feature map channels
	int    BatchSize=0;             // input-output batch size
	float *dataout=0;               // pointer to output data (CPU)
	float *datain=0;                // pointer to input data ( CPU)
	float *dout=0;                  // pointer to output data ( GPU)
	float *din=0;                   // pointer to input data ( GPU)
	float *h_convWeights=0;         // pointer to filter data ( CPU)
	
	


	// optional bias
	hipdnnTensorDescriptor_t inTensor, outTensor, outTensorR, BiasTensor; // cudnn specific Tensors
	hipdnnFilterDescriptor_t FilterDesc;                                  // cudnn Specific Descriptors

	hipdnnConvolutionDescriptor_t convDesc;                               // cudnn Specific Descriptor
	hipdnnConvolutionFwdAlgo_t convAlgo;                                  // cudnn Specific variable for algorithm type

	// a handle
	// hipdnnHandle_t   cudnnHandle;                                         // pointer  to a cudnn Handle
	
	size_t workspaceSize;                                               // workspace size used by the operations



	float *convWeights  ;                                                 // pointers to filter weights (GPU)
	void *workspace = nullptr;                                            // pointers to cudnn workspaceSize
	
	
	// set to arbitrary large value
	size_t workspace_limit_bytes = 32*1024*1024;
	
	//-----------------------------------------------
	//						 print cudnn version
	//-----------------------------------------------
	
	printf("cudnn version: %u\n", hipdnnGetVersion());



	//--------------------------------------------------------
	//          cudnn set stream
	//--------------------------------------------------------
	hipdnnSetStream( cudnnHandle, cudaStream);
	
	//-----------------------------------------------
	// [1] Setting filter sizes. Assume that we have an input image of three channels, batch size 1, and a 128 filters
	//-----------------------------------------------
	BatchSize    =1;
    in_channels  =3;
	out_channels =128; // assume that we have 128 filters of 3 x 3 size. 
	in_height    =224;
	in_width     = 224;
	kernel_size = 3;  // assume that filter width ==height
	stepsize    = 1;
	padd        = 0;
	dilation_h  = 1;
	dilation_w  = 1;
	
	

	
	
	

	//--------------------------------------------------------------------
	//                Setup Tensors, Filter etc. Create and initialize
	//--------------------------------------------------------------------



 
	CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inTensor));
 
	CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inTensor,
		HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT,
		BatchSize, in_channels,
		in_height, in_width));
	//}
	CHECK_CUDNN(hipdnnCreateFilterDescriptor(&FilterDesc));
	// set dimensions
	CHECK_CUDNN(hipdnnSetFilter4dDescriptor(FilterDesc,
		HIPDNN_DATA_FLOAT,
		HIPDNN_TENSOR_NCHW,
		out_channels, // number of filters
		in_channels,
		kernel_size,
		kernel_size));

	CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
	// set dimensions
	CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
		padd, padd,
		stepsize, stepsize,         // stride
		dilation_h, dilation_w,     // These variables define dilation. supported from Cudnn 6.0 and later
		HIPDNN_CROSS_CORRELATION,    //HIPDNN_CONVOLUTION
		HIPDNN_DATA_FLOAT       ));  // operation mode- precision 
	
	// Find dimension of convolution output
	CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc,
		inTensor,
		FilterDesc,
		&BatchSize, &out_channels, &out_height, &out_width));

	std::cout << "output width" << out_width << "output height" << out_height << std::endl;

 

	// Output Tensor
	CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outTensor));
	// set dimensions [Library Version]
	CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outTensor,
		HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT,
		BatchSize, out_channels,
		out_height, out_width));
    CHECK_CUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
		inTensor,
		FilterDesc,
		convDesc,
		outTensor,
		HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, //HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, //HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, //HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, //HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
		workspace_limit_bytes,
		&convAlgo));
 
	// For that given algorithm  find the WorkSpace Size in bytes
	// In general we set the size as the maximum needed across operations
	CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
		inTensor,
		FilterDesc,
		convDesc,
		outTensor,
		convAlgo,
		&workspaceSize));
 
		std::cout << "Workspace size in bytes is " << workspaceSize << std::endl;
 
 
 // Note on memoty allocation. Recently, cudnn introduced managed memory allocation. this provides flexibility as it follows the shared memory model
 // however it is expected to have worst performance than custom mallocs

	// Allocate space for input output, convolutional filters and bias
	hipMalloc(&din, sizeof(float) * BatchSize * in_channels* in_height *in_width); // data in
	 
	
	 hipMalloc(&dout, sizeof(float) * BatchSize* out_channels * out_height * out_width); // data out (after conv)
	 hipMalloc(&convWeights, sizeof(float) * BatchSize* in_channels* out_channels * kernel_size * kernel_size); // filter coeffs
 	 
	if (workspaceSize > 0)
		 (hipMalloc(&workspace, workspaceSize)); 
  
    // Allocate space on host. hipHostMalloc takes a double pointer and allocates pinned memory.
	 hipHostMalloc(&datain, in_channels*in_width*in_height*sizeof(float));
	 hipHostMalloc(&dataout, out_channels*out_width*out_height*sizeof(float));
	 hipHostMalloc(&h_convWeights, BatchSize* in_channels* out_channels * kernel_size * kernel_size*sizeof(float));
	 
	 
	 
	 //-----------------------------------------------------------
	 // TODO: Load data to datain from DISK and then copy them to din.
	 // TODO: Load filter weights to h_convWeights from DISK and then copy them to convWeights.
	 // din is the cuda memory and what we want to do is first load data into cpu memory (datain) and then copy them to cuda (din)
	 // This memcpy MUST be performed with hipMemcpy
	 // ADD your code here to load data from CPU to cuda
	 //
	 // FREAD == > h_convWeights and then hipMemcpy (h_convWeights ==> convWeights)
	 // FREAD == > datain and then hipMemcpy (datain ==> din)
	 // 
	 //
	
 
		//create timer.
		std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

          //The following operation is a BLOCKING OPERATION. therefore, you can use cpu timer. However, it is more accurate to use cuda time (event)		 
		for (int i = 0; i < ITERS; i++) {
			
			const float alpha1 = 1.0f, beta1 = 1.0f, beta0 = 0.0f;

		 
		CHECK_CUDNN(hipdnnConvolutionForward(cudnnHandle,
			(void*)&alpha1,
			inTensor, (void*)din,  //data (Tensor and pointer to GPU array)
			FilterDesc, (void*)convWeights,   //Filter (Descriptor and pointer to GPU array)
			convDesc,                 //Convolution (Descriptor)
			convAlgo,
			workspace,
			workspaceSize,
			(void*)&beta0,
			outTensor, (void*)dout));
 
		}

		std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
		printf("Time: %2.3f\n",
			double(std::chrono::duration_cast<std::chrono::nanoseconds>(t2-t1).count())
			/ (1e6 * float(ITERS))
		);
 
 
	 
 
	return 0;
}
 